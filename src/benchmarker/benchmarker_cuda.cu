#include "hip/hip_runtime.h"
#include "benchmarker.hpp"
#include <nanovdb/util/Ray.h>
#include <vector>

#include <chrono>

__global__ void kernel_raytracing(nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_level_set,
                                  Benchmarker::NVDB_RayT *rays, FP_Type *time_results,
                                  nanovdb::Coord *result_coords, size_t n_rays, int load_factor)
{
  unsigned int n_threads = blockDim.x * gridDim.x;
  unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  auto acc = d_level_set->tree().getAccessor();
  nanovdb::Coord ijk;
  FP_Type value;

  for(unsigned int reps = 0; reps < load_factor; reps++)
  {
    for (unsigned int i = thread_id; i < n_rays; i += n_threads)
    {
      nanovdb::ZeroCrossing(rays[i], acc, result_coords[i], value, time_results[i]);
    }
  }
}

void Benchmarker::run_nanoVDB_GPU(nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> &level_set, std::vector<Benchmarker::OVBD_Vec3T> const & reference_solution,
                                  size_t n_rays)
{

  // GPU
  int grid_size = (size_t)options["gpu_grid_size"].as<int>();
  int block_size = (size_t)options["gpu_block_size"].as<int>();
  int load_factor = (size_t)options["gpu_load_factor"].as<int>();

  size_t bytes = 0;
  nanovdb::FloatGrid *grid_handle = level_set.grid<FP_Type>();

  // Init Grid on GPU
  level_set.deviceUpload(); // TODO: move outside so it wont be called every time
  nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_grid_handle = level_set.deviceGrid<FP_Type>();
  if (!d_grid_handle)
    throw std::runtime_error("GridHandle does not contain a valid device grid");

  // Init rays on GPU
  bytes = sizeof(NVDB_RayT) * n_rays;
  std::vector<NVDB_RayT> rays =
      generate_rays<NVDB_GridT, NVDB_RayT>(*grid_handle, n_rays); // TODO: change to levelset


  NVDB_RayT *d_rays;
  hipMalloc(&d_rays, bytes);
  hipMemcpy(d_rays, rays.data(), bytes, hipMemcpyHostToDevice);

  // Allocate Results on GPU
  bytes = sizeof(OVBD_Vec3T) * n_rays;
  std::vector<FP_Type> result_times(n_rays);
  FP_Type *d_result_times;
  hipMalloc(&d_result_times, bytes);
  bytes = sizeof(nanovdb::Coord) * n_rays;
  std::vector<nanovdb::Coord> result_coords(n_rays);
  nanovdb::Coord *d_result_coords;
  hipMalloc(&d_result_coords, bytes);

  // Start Benchmark
  Timer timer;
  hipDeviceSynchronize();
  timer.reset();

  kernel_raytracing<<<grid_size, block_size>>>(d_grid_handle, d_rays, d_result_times,
                                               d_result_coords, n_rays, load_factor);
  hipDeviceSynchronize();

  double time = timer.get();
  double adjusted_time = time / (double) load_factor;

  // Transfer results back to CPU
  hipMemcpy(result_times.data(), d_result_times, sizeof(result_times[0]) * n_rays,
             hipMemcpyDeviceToHost);

  hipMemcpy(result_coords.data(), d_result_coords, sizeof(result_coords[0]) * n_rays,
             hipMemcpyDeviceToHost);

  std::vector<Benchmarker::OVBD_Vec3T> wResults = indexToWorld(*grid_handle, result_coords);
  analyze_results(wResults, reference_solution);

  if(options.count("calculate-error"))
  {
    std::cout << "Calculating Error for NanoVDB on GPU" << std::endl;
    calculate_error(wResults, result_times);
  }

  // free up GPU Allocations
  hipFree(d_rays);
  hipFree(d_result_coords);
  hipFree(d_result_times);

  PLOG_INFO << "NanoVDB on GPU Finished in " << time << "s (" << (double)n_rays / (1e6 * adjusted_time)
            << " MRays/s)" << std::endl;

  write_results(result_file, "NanoVDB_GPU", n_rays, adjusted_time, grid_size, block_size,
                options["gpu_price"].as<double>(), options["gpu_power"].as<double>());
}
