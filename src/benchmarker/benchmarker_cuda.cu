#include "hip/hip_runtime.h"
#include "benchmarker.hpp"
#include <nanovdb/util/Ray.h>
#include <vector>

__global__ void kernel_raytracing(nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_level_set,
                                  Benchmarker::NVDB_RayT *rays, FP_Type *time_results,
                                  nanovdb::Coord *result_coords, size_t n_rays)
{
  unsigned int n_threads = blockDim.x * gridDim.x;
  unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  auto acc = d_level_set->tree().getAccessor();
  nanovdb::Coord ijk;
  FP_Type value;

  for (unsigned int i = thread_id; i < n_rays; i += n_threads)
  {
    nanovdb::ZeroCrossing(rays[i], acc, result_coords[i], value, time_results[i]);
  }
}

void Benchmarker::run_nanoVDB_GPU(nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> &level_set,
                                  size_t n_rays)
{
  size_t bytes = 0;
  nanovdb::FloatGrid *grid_handle = level_set.grid<FP_Type>();

  std::vector<OVBD_Vec3T> reference_intersections =
      calculate_reference_solution<OVBD_Vec3T>(n_rays, options["r1"].as<double>());

  // Init Grid on GPU
  level_set.deviceUpload(); // TODO: move outside so it wont be called every time
  nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_grid_handle = level_set.deviceGrid<FP_Type>();
  if (!d_grid_handle)
    throw std::runtime_error("GridHandle does not contain a valid device grid");

  // Init rays on GPU
  bytes = sizeof(NVDB_RayT) * n_rays;
  std::vector<NVDB_RayT> rays =
      generate_rays<NVDB_GridT, NVDB_RayT>(*grid_handle, n_rays); // TODO: change to levelset
  NVDB_RayT *d_rays;
  hipMalloc(&d_rays, bytes);
  hipMemcpy(d_rays, rays.data(), bytes, hipMemcpyHostToDevice);

  // Allocate Results on GPU
  bytes = sizeof(OVBD_Vec3T) * n_rays;
  std::vector<FP_Type> result_times(n_rays);
  FP_Type *d_result_times;
  hipMalloc(&d_result_times, bytes);
  bytes = sizeof(nanovdb::Coord) * n_rays;
  std::vector<nanovdb::Coord> result_coords(n_rays);
  nanovdb::Coord *d_result_coords;
  hipMalloc(&d_result_coords, bytes);

  // Start Benchmark
  Timer timer;
  timer.reset();
  kernel_raytracing<<<grid_size, block_size>>>(d_grid_handle, d_rays, d_result_times,
                                               d_result_coords, n_rays);
  hipDeviceSynchronize();
  double time = timer.get();

  // Transfer results back to CPU
  hipMemcpy(result_times.data(), d_result_times, sizeof(result_times[0]) * n_rays,
             hipMemcpyDeviceToHost);

  hipMemcpy(result_coords.data(), d_result_coords, sizeof(result_coords[0]) * n_rays,
             hipMemcpyDeviceToHost);

  auto wResults = indexToWorld(*grid_handle, result_coords);
  analyze_results(wResults, reference_intersections);

  // free up GPU Allocations
  hipFree(d_rays);
  hipFree(d_result_coords);
  hipFree(d_result_times);

  PLOG_INFO << "NanoVDB on GPU Finished in " << time << "s (" << (double)n_rays / (1e6 * time)
            << " MRays/s)" << std::endl;

  write_results(result_file, "NanoVDB_GPU", n_rays, time, grid_size, block_size,
                options["cpu_price"].as<double>());
}
