#include "hip/hip_runtime.h"
#include "benchmarker.hpp"

#include <nanovdb/util/Ray.h>

#include <vector>

// TODO: switch to common definition of
using FP_Type = float;
using RayT = nanovdb::Ray<FP_Type>;
using Vec3T = nanovdb::Vec3<FP_Type>;

// TODO: rename fucntion
__global__ void run_cuda(nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_level_set, RayT *rays,
                         size_t n_rays)
{

  unsigned int n_threads = blockDim.x * gridDim.x;
  unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  auto acc = d_level_set->tree().getAccessor();
  nanovdb::Coord ijk;
  FP_Type t0 = 0;
  FP_Type v;

  for (unsigned int i = thread_id; i < n_rays; i += n_threads)
  {
    nanovdb::ZeroCrossing(rays[i], acc, ijk, v, t0);
    assert(t0 > 0); // TODO: replace with proper result verification
  }
}

/**
 * @brief Wrapper for launching CUDA Kernels.
 *  
 * 
 * @tparam CALLABLE 
 * @tparam Arg 
 * @param grid_size 
 * @param bock_size 
 * @param callable 
 * @param args 
 */
template <class CALLABLE, class... Arg>
void Benchmarker::launch_kernel(size_t grid_size, size_t bock_size, CALLABLE &&callable,
                                Arg &&... args)
{
  callable<<<grid_size, bock_size>>>(std::forward<Arg>(args)...);
}

void Benchmarker::run_nanoVDB_GPU(nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> &grid_handle,
                                  size_t n_rays)
{
  using FP_Type = float;
  using RayT = nanovdb::Ray<FP_Type>;
  using Vec3T = nanovdb::Vec3<FP_Type>;

  size_t grid_size = 256;
  size_t block_size = 256;
  size_t bytes = 0;

  FP_Type sphere_radius_outer = 5; // TODO: replace with optionsd argument

  assert(n_rays > 0);

  std::vector<Vec3T> reference_solutions =
      calculate_reference_solution<Vec3T>(n_rays, sphere_radius_outer);

  // Init Grid on GPU
  grid_handle.deviceUpload(); // TODO: move outside so it wont be called every time
  nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_grid_handle = grid_handle.deviceGrid<FP_Type>();
  if (!d_grid_handle)
    throw std::runtime_error("GridHandle does not contain a valid device grid");

  // Init rays on GPU
  bytes = sizeof(RayT) * n_rays;
  std::vector<RayT> rays = generate_rays<RayT>(n_rays);
  RayT *d_rays;
  hipMalloc(&d_rays, bytes);
  hipMemcpy(d_rays, rays.data(), bytes, hipMemcpyHostToDevice);

  // Run Benchmark
  std::vector<Vec3T> calculated(n_rays, Vec3T(0, 0, 0)); // results

  Timer timer;
  timer.reset();
  launch_kernel(grid_size, block_size, run_cuda, d_grid_handle, d_rays, n_rays);
  hipDeviceSynchronize();

  double time = timer.get();
  PLOG_INFO << "NanoVDB on GPU Finished in " << time << "s (" << (double)n_rays / (1e6 * time)
            << " MRays/s)" << std::endl;

  hipFree(d_rays);
}
