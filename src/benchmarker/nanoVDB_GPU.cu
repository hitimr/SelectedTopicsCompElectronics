#include "hip/hip_runtime.h"
#include "benchmarker.hpp"
#include "nanoVDB_GPU.hpp"

#include <nanovdb/util/Ray.h>

#include <vector>

// TODO: switch to common definition of
using FP_Type = float;
using RayT = nanovdb::Ray<FP_Type>;
using Vec3T = nanovdb::Vec3<FP_Type>;

// TODO: rename fucntion
__global__ void run(nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_level_set, RayT *rays,
                    size_t n_rays)
{

  unsigned int n_threads = blockDim.x * gridDim.x;
  unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  auto acc = d_level_set->tree().getAccessor();
  nanovdb::Coord ijk;
  FP_Type t0 = 0;
  FP_Type v;

  for (unsigned int i = thread_id; i < n_rays; i += n_threads)
  {
        nanovdb::ZeroCrossing(rays[i], acc, ijk, v, t0);
        assert(t0 > 0); // TODO: replace with proper result verification

  }
}

void run_nanoVDB_GPU(nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> &grid_handle, size_t n_rays)
{
  using FP_Type = float;
  using RayT = nanovdb::Ray<FP_Type>;
  using Vec3T = nanovdb::Vec3<FP_Type>;

  size_t grid_size = 256;
  size_t block_size = 256;
  size_t bytes = 0;

  FP_Type sphere_radius_outer = 5; // TODO: replace with optionsd argument

  assert(n_rays > 0);

  std::vector<Vec3T> reference_solutions =
      calculate_reference_solution<Vec3T>(n_rays, sphere_radius_outer);

  // Init Grid on GPU
  grid_handle.deviceUpload(); // TODO: move outside so it wont be called every time
  nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_grid_handle = grid_handle.deviceGrid<FP_Type>();
  if (!d_grid_handle)
    throw std::runtime_error("GridHandle does not contain a valid device grid");

  // Init rays on GPU
  bytes = sizeof(RayT) * n_rays;
  std::vector<RayT> rays = generate_rays<RayT>(n_rays);
  RayT *d_rays;
  hipMalloc(&d_rays, bytes);
  hipMemcpy(d_rays, rays.data(), bytes, hipMemcpyHostToDevice);

  // Run Benchmark
  std::vector<Vec3T> calculated(n_rays, Vec3T(0, 0, 0)); // results

  Timer timer;
  timer.reset();
  run<<<grid_size, block_size>>>(d_grid_handle, d_rays, n_rays);
  hipDeviceSynchronize();

  double time = timer.get();
  PLOG_INFO << "NanoVDB on GPU Finished in " << time << "s (" << (double)n_rays / (1000 * time)
            << " kRays/s)" << std::endl;

  hipFree(d_rays);
}
