#include "hip/hip_runtime.h"
#include "benchmarker.hpp"
#include "nanoVDB_GPU.hpp"

#include <nanovdb/util/Ray.h>

#include <vector>

__global__ void run(nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_level_set, size_t n_rays)
{

  auto acc = d_level_set->tree().getAccessor();
  nanovdb::Coord ijk;
  FP_Type t0;
  FP_Type v;
}

void run_nanoVDB_GPU(nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> &grid_handle, size_t n_rays)
{
  using FP_Type = float;
  using RayT = nanovdb::Ray<FP_Type>;
  using Vec3T = nanovdb::Vec3<FP_Type>;

  FP_Type sphere_radius_outer = 5; // TODO: replace with optionsd argument

  assert(n_rays > 0);
  PLOG_INFO << "Running NanoVDP on GPU benchmark for " << n_rays << " Rays" << std::endl;

  std::vector<RayT> rays = generate_rays<RayT>(n_rays);
  std::vector<Vec3T> reference_solutions =
      calculate_reference_solution<Vec3T>(n_rays, sphere_radius_outer);

  // Send data to GPU
  grid_handle.deviceUpload(); // TODO: move outside so it wont be called every time
  nanovdb::Grid<nanovdb::NanoTree<FP_Type>> * d_grid_handle = grid_handle.deviceGrid<FP_Type>();
  if (!d_grid_handle)
    throw std::runtime_error("GridHandle does not contain a valid device grid");

  // Run Benchmark
  std::vector<Vec3T> calculated(n_rays, Vec3T(0, 0, 0)); // results
  run<<<256, 256>>>(d_grid_handle, n_rays);
}
