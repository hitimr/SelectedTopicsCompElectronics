#include "hip/hip_runtime.h"
#include "benchmarker.hpp"

#include <nanovdb/util/Ray.h>

#include <vector>

// TODO: switch to common definition of
using FP_Type = float;
using RayT = nanovdb::Ray<FP_Type>;
using Vec3T = nanovdb::Vec3<FP_Type>;

// TODO: rename fucntion
__global__ void run_cuda(nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_level_set, RayT *rays,
                         FP_Type *time_results, nanovdb::Coord *result_coords, size_t n_rays)
{

  unsigned int n_threads = blockDim.x * gridDim.x;
  unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  auto acc = d_level_set->tree().getAccessor();
  nanovdb::Coord ijk;
  FP_Type value;

  for (unsigned int i = thread_id; i < n_rays; i += n_threads)
  {
    nanovdb::ZeroCrossing(rays[i], acc, result_coords[i], value, time_results[i]);
  }
}

void Benchmarker::run_nanoVDB_GPU(nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> &grid_handle,
                                  size_t n_rays)
{
  using FP_Type = float;
  using RayT = nanovdb::Ray<FP_Type>;
  using Vec3T = nanovdb::Vec3<FP_Type>;

  size_t grid_size = 256;
  size_t block_size = 256;
  size_t bytes = 0;

  FP_Type sphere_radius_outer = 5; // TODO: replace with optionsd argument

  assert(n_rays > 0);

  std::vector<Vec3T> reference_solutions =
      calculate_reference_solution<Vec3T>(n_rays, sphere_radius_outer);

  // Init Grid on GPU
  grid_handle.deviceUpload(); // TODO: move outside so it wont be called every time
  nanovdb::Grid<nanovdb::NanoTree<FP_Type>> *d_grid_handle = grid_handle.deviceGrid<FP_Type>();
  if (!d_grid_handle)
    throw std::runtime_error("GridHandle does not contain a valid device grid");

  // Init rays on GPU
  bytes = sizeof(RayT) * n_rays;
  std::vector<RayT> rays = generate_rays<RayT>(n_rays);
  RayT *d_rays;
  hipMalloc(&d_rays, bytes);
  hipMemcpy(d_rays, rays.data(), bytes, hipMemcpyHostToDevice);

  // Allocate Results on GPU
  bytes = sizeof(Vec3T) * n_rays;
  std::vector<FP_Type> result_times(n_rays);
  FP_Type *d_result_times;
  hipMalloc(&d_result_times, bytes);

  bytes = sizeof(nanovdb::Coord) * n_rays;
  std::vector<nanovdb::Coord> result_coords(n_rays);
  nanovdb::Coord *d_result_coords;
  hipMalloc(&d_result_coords, bytes);

  // Start Bennchmark
  Timer timer;
  timer.reset();
  run_cuda<<<grid_size, block_size>>>(d_grid_handle, d_rays, d_result_times, d_result_coords,
                                      n_rays);
  hipDeviceSynchronize();
  double time = timer.get();

  PLOG_INFO << "NanoVDB on GPU Finished in " << time << "s (" << (double)n_rays / (1e6 * time)
            << " MRays/s)" << std::endl;

  // Transfer results back to CPU
  hipMemcpy(result_times.data(), d_result_times, sizeof(result_times[0]) * n_rays,
             hipMemcpyDeviceToHost);

  hipMemcpy(result_coords.data(), d_result_coords, sizeof(result_coords[0]) * n_rays,
             hipMemcpyDeviceToHost);

  auto *h_grid = grid_handle.grid<FP_Type>();
  std::vector<nanovdb::Vec3<FP_Type>> result_intersections(n_rays);
  for (size_t i = 0; i < n_rays; i++)
  {
    result_intersections[i] = h_grid->indexToWorldF<Vec3T>(result_coords[i].asVec3s());
  }

  hipFree(d_rays);
  hipFree(d_result_coords);
  hipFree(d_result_times);
}
